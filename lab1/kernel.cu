#define TILE_WIDTH 16

// System includes
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <cstring> // Added for strcmp
#include <ctime>
#include <chrono>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Function to compare result matrices from CPU and GPU
bool compareMatrices(float* C, float* D, int size) 
{
    for (int i = 0; i < size; ++i) {
        float error = std::abs(C[i] - D[i]);
        if (error > 1e-3) {
            return false;
        }
    }
    return true;
}

// Function to initialize matrices A and B
void initializeMatrices(float* matrix, int size) 
{
    std::srand(static_cast<unsigned>(std::time(nullptr)));
    for (int i = 0; i < size; ++i) 
    {
        // Generate a random float number between 0 and 1
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// CPU implementation
void matrixMultiplication(float* A, float* B, float* D, int w) 
{
    for (int i = 0; i < w; ++i) {
        for (int j = 0; j < w; ++j) {
            float sum = 0.0;
            for (int k = 0; k < w; ++k) {
                sum += A[i * w + k] * B[k * w + j];
            }
            D[i * w + j] = sum;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
// Do not modify code above this line
////////////////////////////////////////////////////////////////////////////////////////////////////////

// GPU implementation
// Implement this kernel function
// A & B are addresses on the host for input matrices, C is the address on the host for output matrix
// matrixWidth is the width of matrices for which matrix multiplication is being performed
__global__ void MatrixMulCUDA(float* C, float* A, float* B, int matrixWidth)
{
	__shared__ float A_tile[TILE_WIDTH][TILE_WIDTH];
	__shared__ float B_tile[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float C_accum = 0.0;

    for (int i = 0; i < matrixWidth / TILE_WIDTH; i++) {
        int a_index = row * matrixWidth + i * TILE_WIDTH + tx;
        int b_index = (i * TILE_WIDTH + ty) * matrixWidth + col;

        A_tile[ty][tx] = A[a_index];
        B_tile[ty][tx] = B[b_index];

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) {
            C_accum += A_tile[ty][k] * B_tile[k][tx];
        }

        __syncthreads();
    }

    C[row * matrixWidth + col] = C_accum;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////
// Do not modify code below this line
////////////////////////////////////////////////////////////////////////////////////////////////////////

/**
 * Program main
 */
int main(int argc, char** argv) 
{
    if (argc != 3 || strcmp(argv[1], "-m") != 0) 
    {
        std::cout << "Usage: ./a.out -m <matrix width>" << std::endl;
        return -1;
    }

    int matrixWidth = atoi(argv[2]);
    
    int matrixSize = matrixWidth * matrixWidth;

    float *A, *B, *C, *D;
    hipMallocManaged(&A, matrixSize * sizeof(float));
    hipMallocManaged(&B, matrixSize * sizeof(float));
    hipMallocManaged(&C, matrixSize * sizeof(float));
    hipMallocManaged(&D, matrixSize * sizeof(float));

    initializeMatrices(A, matrixSize);
    initializeMatrices(B, matrixSize);

    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
    dim3 gridSize(matrixWidth / TILE_WIDTH, matrixWidth / TILE_WIDTH);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float gpu_time = 0;
    float cpu_time = 0;

    hipEventRecord(start);
    // Launch the kernel
    MatrixMulCUDA<<<gridSize, blockSize>>>(C, A, B, matrixWidth);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&gpu_time, start, stop);

    auto start_time = std::chrono::high_resolution_clock::now();
    // Perform matrix multiplication on CPU and store in D
    matrixMultiplication (A, B, D, matrixWidth);    
    auto end_time = std::chrono::high_resolution_clock::now();
    cpu_time = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count();
    cpu_time = cpu_time / 1000;

    // ... (Perform matrix multiplication on CPU and store in D)

    // Compare matrices C and D
    bool matricesMatch = compareMatrices(C, D, matrixSize);
    
    if (matricesMatch) {
        printf("SUCCESS!\n");
        printf("CPU Matrix Multiply Time (ms) : %f \n", cpu_time);
        printf("GPU Matrix Multiply Time (ms) : %f \n", gpu_time);
	printf("Speedup: %f \n", cpu_time/gpu_time);
    } else {
        printf("ERROR!\n");
    }

    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(D);

    return 0;    
}
